#include "hip/hip_runtime.h"
#include "mpm_solver3d.h"
#include "collision_object3d.h"
#include "create_file.h"
#include "cuda_utils.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/tabulate.h>
#include <iostream>
#include <limits>
#include <openvdb/openvdb.h>
#include <openvdb/io/Stream.h>

namespace chains {

struct InstantiateCollocatedGridData3DWithIndex {
    Eigen::Vector3i resolution;

    InstantiateCollocatedGridData3DWithIndex(Eigen::Vector3i gridResolution) : resolution(gridResolution) {}

    __host__ __device__
    CollocatedGridData3D operator()(const int& idx) {
        return CollocatedGridData3D(
            Eigen::Vector3i(
                idx % resolution(0),
                ((idx / resolution(0)) % resolution(1)),
                ((idx / resolution(0)) / resolution(1))
            )
        );
    }
};

MPMSolver3D::MPMSolver3D(
    const thrust::host_vector<MaterialPoint3D> &particles,
    Eigen::Vector3d gridOrigin,
    Eigen::Vector3i gridResolution,
    double gridStride,
    double gridBoundaryFrictionCoefficient,
    double blendCoefficient,
    InterpolationType interpolationType,
    double deltaTime
) : _blend_coefficient(blendCoefficient), _delta_time(deltaTime)
{
    std::cout << "[INFO] copying particles to device" << std::endl;
    // Build particles
    _particles.resize(particles.size());
    thrust::copy(particles.begin(), particles.end(), _particles.begin());
    CUDA_CHECK_LAST_ERROR();

    // Build GridSettings on host and device
    _host_grid_settings = (Grid3DSettings*)malloc(sizeof(Grid3DSettings));
    *_host_grid_settings= Grid3DSettings(
        gridOrigin,
        gridResolution,
        gridStride,
        gridBoundaryFrictionCoefficient
    );
    //  Copy to device
    CUDA_CHECK(hipMalloc((void**) &_grid_settings, sizeof(Grid3DSettings)));
    CUDA_CHECK(hipMemcpy(_grid_settings, _host_grid_settings, sizeof(Grid3DSettings), hipMemcpyHostToDevice));

    // Build interpolator on host and device
    _host_interpolator = (Interpolator3D*)malloc(sizeof(Interpolator3D));
    *_host_interpolator = Interpolator3D(interpolationType);
    //  Copy to device
    CUDA_CHECK(hipMalloc((void**) &_interpolator, sizeof(Interpolator3D)));
    CUDA_CHECK(hipMemcpy(_interpolator, _host_interpolator, sizeof(Interpolator3D), hipMemcpyHostToDevice));

    std::cout << "[INFO] building grid of "
              << gridResolution(0) << " x " << gridResolution(1) << " x " << gridResolution(2)
              << " on device" << std::endl;
    // Build grid
    int grid_size = gridResolution(0)*gridResolution(1)*gridResolution(2);
    _grid.resize(grid_size);
    thrust::tabulate(
        thrust::device,
        _grid.begin(),
        _grid.end(),
        InstantiateCollocatedGridData3DWithIndex(gridResolution)
    );
    CUDA_CHECK_LAST_ERROR();

    _enable_particles_collision = false;

    std::cout << "[INFO] initializing particle initial volume" << std::endl;
    // IMPORTANT: Compute the initialize volume of each particle
    initialize();
}

MPMSolver3D::~MPMSolver3D() {
    CUDA_CHECK(hipFree(_grid_settings));
    CUDA_CHECK(hipFree(_interpolator));
}

void MPMSolver3D::simulateOneStep() {
    double dt = _delta_time;    // TODO: CFL

    // New grid for this step
    resetGrid();

    // PIC: Transfer mass, velocity and elastic force
    //  NOTICE: this includes computing explicit grid forces
    particlesToGrid();

    computeExternalForces();

    updateGridVelocities(dt);
    gridCollision(dt);
    solveLinearSystem(dt);

    // PIC: Transfer velocity
    //  NOTICE: this includes updating paticles' deformation gradient 
    gridToParticles(dt, _blend_coefficient);

    if (_enable_particles_collision) {
        particlesCollision(dt);
    }
 
    advectParticles(dt);
}

void MPMSolver3D::switchIfEnableParticlesCollision() {
    _enable_particles_collision = _enable_particles_collision==true? false : true;
}

void MPMSolver3D::initialize() {
    CollocatedGridData3D* grid_ptr = thrust::raw_pointer_cast(&_grid[0]);
    Grid3DSettings* grid_settings_ptr = _grid_settings; 
    Interpolator3D* interpolator_ptr = _interpolator; 

    auto P2GMass = [=] __device__ (MaterialPoint3D& mp) {
        Eigen::Vector3d ori = grid_settings_ptr->origin;
        Eigen::Vector3i res = grid_settings_ptr->resolution;
        double h = grid_settings_ptr->stride;
        double rng = interpolator_ptr->_range;

        Eigen::Vector3d mp_pos = mp.position;
    
        // Grid vertex range inside the interpolation kernel
        int index_l[3], index_u[3];
        for (int i = 0; i < 3; i ++) {
            index_l[i] = ceil(mp_pos(i)/h - rng);
            index_u[i] = floor(mp_pos(i)/h + rng);
        }

        for (int x = index_l[0]; x <= index_u[0]; x ++) {
            for (int y = index_l[1]; y <= index_u[1]; y ++) {
                for (int z = index_l[2]; z <= index_u[2]; z ++) {
                     if (x < 0 || x >= res(0)
                      || y < 0 || y >= res(1)
                      || z < 0 || z >= res(2)) continue; // Ensure the vertex is inside the grid
                    // 1D index to access grid data
                    int gd_index = x + res(0)*(y + res(1)*z);
                    // Grid vertex world position
                    Eigen::Vector3d gd_pos = ori + h * Eigen::Vector3d(x, y, z);

                    double w = interpolator_ptr->weight3D(mp_pos, gd_pos, h);

                    // PIC-FLIP
                    //  mass
                    double m_ip = mp.mass * w;
                    atomicAdd(&(grid_ptr[gd_index].mass), m_ip);
                }
            }
        }
    };

    auto G2PVolume = [=] __device__ (MaterialPoint3D& mp) {
        Eigen::Vector3d ori = grid_settings_ptr->origin;
        Eigen::Vector3i res = grid_settings_ptr->resolution;
        double h = grid_settings_ptr->stride;
        double rng = interpolator_ptr->_range;

        Eigen::Vector3d mp_pos = mp.position;
    
        // Grid vertex range inside the interpolation kernel
        int index_l[3], index_u[3];
        for (int i = 0; i < 3; i ++) {
            index_l[i] = ceil(mp_pos(i)/h - rng);
            index_u[i] = floor(mp_pos(i)/h + rng);
        }

        double mp_density = 0.0;
        double inv_cell_vol = 1.0 / (h*h*h);

        for (int x = index_l[0]; x <= index_u[0]; x ++) {
            for (int y = index_l[1]; y <= index_u[1]; y ++) {
                for (int z = index_l[2]; z <= index_u[2]; z ++) {
                     if (x < 0 || x >= res(0)
                      || y < 0 || y >= res(1)
                      || z < 0 || z >= res(2)) continue; // Ensure the vertex is inside the grid
                    // 1D index to access grid data
                    int gd_index = x + res(0)*(y + res(1)*z);
                    // Grid vertex world position
                    Eigen::Vector3d gd_pos = ori + h * Eigen::Vector3d(x, y, z);

                    double w = interpolator_ptr->weight3D(mp_pos, gd_pos, h);

                    // PIC-FLIP
                    double m_ip = grid_ptr[gd_index].mass * w;
                    //  accumulate density
                    mp_density += m_ip * inv_cell_vol;
                }
            }
        }

        // Set the initial volume of particle
        mp.volume0 = mp.mass / mp_density;
    };

    // Transfer mass from particles to grid
    thrust::for_each(thrust::device, _particles.begin(), _particles.end(), P2GMass);
    CUDA_CHECK_LAST_ERROR();

    // Compute particle volumes and densities back
    thrust::for_each(thrust::device, _particles.begin(), _particles.end(), G2PVolume);
    CUDA_CHECK_LAST_ERROR();
}

void MPMSolver3D::resetGrid() {
    thrust::for_each(
        thrust::device,
        _grid.begin(),
        _grid.end(),
        [=] __device__ (CollocatedGridData3D& gd) {
            gd.reset();
        }
    );
}

void MPMSolver3D::particlesToGrid() {
    CollocatedGridData3D* grid_ptr = thrust::raw_pointer_cast(&_grid[0]);
    Grid3DSettings* grid_settings_ptr = _grid_settings; 
    Interpolator3D* interpolator_ptr = _interpolator; 

    auto P2G = [=] __device__ (MaterialPoint3D& mp) {
        Eigen::Vector3d ori = grid_settings_ptr->origin;
        Eigen::Vector3i res = grid_settings_ptr->resolution;
        double h = grid_settings_ptr->stride;
        double rng = interpolator_ptr->_range;

        Eigen::Vector3d mp_pos = mp.position;
    
        // Grid vertex range inside the interpolation kernel
        int index_l[3], index_u[3];
        for (int i = 0; i < 3; i ++) {
            index_l[i] = ceil(mp_pos(i)/h - rng);
            index_u[i] = floor(mp_pos(i)/h + rng);
        }

        Eigen::Matrix3d vol_stress = -mp.volumeTimesCauchyStress();

        for (int x = index_l[0]; x <= index_u[0]; x ++) {
            for (int y = index_l[1]; y <= index_u[1]; y ++) {
                for (int z = index_l[2]; z <= index_u[2]; z ++) {
                     if (x < 0 || x >= res(0)
                      || y < 0 || y >= res(1)
                      || z < 0 || z >= res(2)) continue; // Ensure the vertex is inside the grid
                    // 1D index to access grid data
                    int gd_index = x + res(0)*(y + res(1)*z);
                    // Grid vertex world position
                    Eigen::Vector3d gd_pos = ori + h * Eigen::Vector3d(x, y, z);

                    double w = interpolator_ptr->weight3D(mp_pos, gd_pos, h);
                    Eigen::Vector3d gradw = interpolator_ptr->weightGradient3D(mp_pos, gd_pos, h);

                    // PIC-FLIP
                    double m_ip = mp.mass * w;
                    Eigen::Vector3d f_ip = vol_stress * gradw;
                    //  mass
                    atomicAdd(&(grid_ptr[gd_index].mass), m_ip);
                    //  velocity
                    //      WARNING: grid velocity is not normalized here
                    atomicAdd(&(grid_ptr[gd_index].velocity(0)), mp.velocity(0)*m_ip);
                    atomicAdd(&(grid_ptr[gd_index].velocity(1)), mp.velocity(1)*m_ip);
                    atomicAdd(&(grid_ptr[gd_index].velocity(2)), mp.velocity(2)*m_ip);
                    //  elastic force
                    atomicAdd(&(grid_ptr[gd_index].force(0)), f_ip(0));
                    atomicAdd(&(grid_ptr[gd_index].force(1)), f_ip(1));
                    atomicAdd(&(grid_ptr[gd_index].force(2)), f_ip(2));
                }
            }
        }
    };

    thrust::for_each(thrust::device, _particles.begin(), _particles.end(), P2G);
}

void MPMSolver3D::computeExternalForces() {
    computeGravityForces();
}

void MPMSolver3D::updateGridVelocities(double deltaTimeInSeconds) {
    thrust::for_each(
        thrust::device,
        _grid.begin(),
        _grid.end(),
        [=] __device__ (CollocatedGridData3D& gd) {
            gd.updateVelocity(deltaTimeInSeconds);
        }
    );
}

void MPMSolver3D::solveLinearSystem(double deltaTimeInSeconds) {
    // TODO
}

void MPMSolver3D::gridToParticles(double deltaTimeInSeconds, double blendCoefficient) {
    CollocatedGridData3D* grid_ptr = thrust::raw_pointer_cast(&_grid[0]);
    Grid3DSettings* grid_settings_ptr = _grid_settings; 
    Interpolator3D* interpolator_ptr = _interpolator; 

    auto computeVelocityAndItsGradient = [=] __device__ (MaterialPoint3D& mp) -> thrust::pair<Eigen::Vector3d, Eigen::Matrix3d> {
        Eigen::Vector3d ori = grid_settings_ptr->origin;
        Eigen::Vector3i res = grid_settings_ptr->resolution;
        double h = grid_settings_ptr->stride;
        double rng = interpolator_ptr->_range;

        Eigen::Vector3d mp_pos = mp.position;
    
        // Grid vertex range inside the interpolation kernel
        int index_l[3], index_u[3];
        for (int i = 0; i < 3; i ++) {
            index_l[i] = ceil(mp_pos(i)/h - rng);
            index_u[i] = floor(mp_pos(i)/h + rng);
        }

        Eigen::Matrix3d vel_grad(Eigen::Matrix3d::Zero());
        Eigen::Vector3d vel_pic(Eigen::Vector3d::Zero());
        Eigen::Vector3d vel_flip(mp.velocity);

        for (int x = index_l[0]; x <= index_u[0]; x ++) {
            for (int y = index_l[1]; y <= index_u[1]; y ++) {
                for (int z = index_l[2]; z <= index_u[2]; z ++) {
                     if (x < 0 || x >= res(0)
                      || y < 0 || y >= res(1)
                      || z < 0 || z >= res(2)) continue; // Ensure the vertex is inside the grid
                    // 1D index to access grid data
                    int gd_index = x + res(0)*(y + res(1)*z);
                    // Grid vertex world position
                    Eigen::Vector3d gd_pos = ori + h * Eigen::Vector3d(x, y, z);

                    double w = interpolator_ptr->weight3D(mp_pos, gd_pos, h);
                    Eigen::Vector3d gradw = interpolator_ptr->weightGradient3D(mp_pos, gd_pos, h);

                    // PIC-FLIP
                    CollocatedGridData3D grid_data = grid_ptr[gd_index];
                    vel_pic += grid_data.velocity_star * w;
                    vel_flip += (grid_data.velocity_star-grid_data.velocity) * w;
                    vel_grad += grid_data.velocity_star * gradw.transpose();
                }
            }
        }

        Eigen::Vector3d vel = blendCoefficient*vel_flip + (1.0-blendCoefficient)*vel_pic;

        return thrust::make_pair(vel, vel_grad);
    };

    thrust::for_each(
        thrust::device,
        _particles.begin(),
        _particles.end(),
        [=] __device__ (MaterialPoint3D& mp) {
            auto vel_pair = computeVelocityAndItsGradient(mp);
            mp.updateDeformationGradient(vel_pair.second, deltaTimeInSeconds);
            mp.velocity = vel_pair.first;
        }
    );
}

void MPMSolver3D::advectParticles(double deltaTimeInSeconds) {
    thrust::for_each(
        thrust::device,
        _particles.begin(),
        _particles.end(),
        [=] __device__ (MaterialPoint3D& mp) {
            mp.updatePosition(deltaTimeInSeconds);
        }
    );
}


void MPMSolver3D::computeGravityForces() {
    auto addGravityForce = [=] __device__ (CollocatedGridData3D& gd) {
        if (gd.mass > std::numeric_limits<double>::epsilon()) {
            gd.force(1) -= 9.80665 * gd.mass; // m/s^2
        }
    };

    thrust::for_each(thrust::device, _grid.begin(), _grid.end(), addGravityForce);
}

void MPMSolver3D::gridCollision(double deltaTimeInSeconds) {
    CollocatedGridData3D* grid_ptr = thrust::raw_pointer_cast(&_grid[0]);
    Grid3DSettings* grid_settings_ptr = _grid_settings; 

    thrust::for_each(
        thrust::device,
        _grid.begin(),
        _grid.end(),
        [=] __device__ (CollocatedGridData3D& gd) {
            Eigen::Vector3d ori = grid_settings_ptr->origin;
            Eigen::Vector3d tar = grid_settings_ptr->target;
            double h = grid_settings_ptr->stride;
            double coeff = grid_settings_ptr->boundary_friction_coefficient;
            gd.velocity_star = applyBoundaryCollision(
                // "Distorted" position of this grid point
                (ori + h*gd.index.cast<double>()) + (deltaTimeInSeconds*gd.velocity_star),
                gd.velocity_star, ori, tar, coeff
            );
        }
    );
}

void MPMSolver3D::particlesCollision(double deltaTimeInSeconds) {
    Grid3DSettings* grid_settings_ptr = _grid_settings; 
    thrust::for_each(
        thrust::device,
        _particles.begin(),
        _particles.end(),
        [=] __device__ (MaterialPoint3D& mp) {
            Eigen::Vector3d ori = grid_settings_ptr->origin;
            Eigen::Vector3d tar = grid_settings_ptr->target;
            double coeff = grid_settings_ptr->boundary_friction_coefficient;
            mp.velocity = applyBoundaryCollision(
                mp.position + deltaTimeInSeconds*mp.velocity,
                mp.velocity, ori, tar, coeff
            );
        }
    );
}

void MPMSolver3D::registerGLBufferWithCUDA(const GLuint buffer) {
    // Enable CUDA to directly map and access the buffer
    CUDA_CHECK(hipGraphicsGLRegisterBuffer(&_cuda_vbo_resource, buffer, cudaGraphicsMapFlagsWriteDiscard));
}

void MPMSolver3D::saveGLBuffer(const GLuint buffer) {
    // Storing the OpenGL buffer ID for future use
    _vbo_buffer = buffer;
}

void MPMSolver3D::updateGLBufferWithCUDA() {
    float4* buf_ptr;
    size_t buf_size;

    CUDA_CHECK(hipGraphicsMapResources(1, &_cuda_vbo_resource, nullptr));

    CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void **)&buf_ptr, &buf_size, _cuda_vbo_resource));

    assert(buf_ptr != nullptr && buf_size >= _particles.size() * sizeof(float4));
    thrust::transform(
        thrust::device,
        _particles.begin(),
        _particles.end(),
        buf_ptr,
        [=] __device__ (MaterialPoint3D& mp) -> float4 {
            return make_float4(
                mp.position(0),
                mp.position(1),
                mp.position(2), 
                1.0
            );
        }
    );

    CUDA_CHECK(hipGraphicsUnmapResources(1, &_cuda_vbo_resource, nullptr));
}

void MPMSolver3D::updateGLBufferByCPU() {
    // Map OpenGL buffer for writing
    glBindBuffer(GL_ARRAY_BUFFER, _vbo_buffer);
    float4 *buf_ptr = (float4*)glMapBuffer(GL_ARRAY_BUFFER, GL_WRITE_ONLY);
    GLint buf_size;
    glGetBufferParameteriv(GL_ARRAY_BUFFER, GL_BUFFER_SIZE, &buf_size);
    
    assert(buf_ptr != nullptr && buf_size >= _particles.size() * sizeof(float4));
    // Copy data from _particles to host particles 
    std::vector<MaterialPoint3D> h_particles(_particles.size());
    thrust::copy(_particles.begin(), _particles.end(), h_particles.begin());
    // Transform and copy data from h_particles to buf_ptr (OpenGL buffer)
    for (size_t i = 0; i < h_particles.size(); i ++) {
        const MaterialPoint3D& mp = h_particles[i];
        buf_ptr[i] = make_float4(
            mp.position(0),
            mp.position(1),
            mp.position(2),
            1.0f
        );
    }

    // Unmap OpenGL buffer
    glUnmapBuffer(GL_ARRAY_BUFFER);
}

void MPMSolver3D::writeToOpenVDB(std::string filePath) {
    // Copy particles to host
    std::vector<MaterialPoint3D> h_particles(_particles.size());
    thrust::copy(_particles.begin(), _particles.end(), h_particles.begin());

    // Create a OpenVDB DoubleGrid
    // openvdb::DoubleGrid::Ptr grid = openvdb::DoubleGrid::create();

    // Build OpenVDB grid
    // Rasterize particles to grid: density
    // for (auto &mp : h_particles) {
    //     Eigen::Vector3d mp_pos = mp.position;

    //     Eigen::Vector3d ori = _host_grid_settings->origin;
    //     Eigen::Vector3i res = _host_grid_settings->resolution;
    //     double h = _host_grid_settings->stride;
    //     double rng = _host_interpolator->_range;

    //     double cell_vol_inv = 1.0 / (h*h*h);

    //     // Grid vertex range inside the interpolation kernel
    //     int index_l[3], index_u[3];
    //     for (int i = 0; i < 3; i ++) {
    //         index_l[i] = ceil(mp_pos(i)/h - rng);
    //         index_u[i] = floor(mp_pos(i)/h + rng);
    //     }

    //     for (int x = index_l[0]; x <= index_u[0]; x ++) {
    //         for (int y = index_l[1]; y <= index_u[1]; y ++) {
    //             for (int z = index_l[2]; z <= index_u[2]; z ++) {
    //                  if (x < 0 || x >= res(0)
    //                   || y < 0 || y >= res(1)
    //                   || z < 0 || z >= res(2)) continue; // Ensure the vertex is inside the grid
    //                 // 1D index to access grid data
    //                 int gd_index = x + res(0)*(y + res(1)*z);
    //                 // Grid vertex world position
    //                 Eigen::Vector3d gd_pos = ori + h * Eigen::Vector3d(x, y, z);

    //                 // PIC-FLIP
    //                 double w = _host_interpolator->weight3D(mp_pos, gd_pos, h);
    //                 // Add density
    //                 openvdb::Coord coord(gd_pos(0), gd_pos(1), gd_pos(2));
    //                 float currentDensity = grid->tree().getValue(coord);
    //                 grid->tree().setValue(coord, currentDensity + mp.mass*w*cell_vol_inv);
    //             }
    //         }
    //     }
    // }

    // // Write grid to file
    // openvdb::io::File file(filePath);
    // openvdb::GridPtrVec grids;
    // grids.push_back(grid);
    // file.write(grids);
    // file.close();
}

}   // namespace chains